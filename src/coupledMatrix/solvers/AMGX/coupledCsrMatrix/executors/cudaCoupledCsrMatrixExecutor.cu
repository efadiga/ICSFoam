#include "hip/hip_runtime.h"
/*---------------------------------------------------------------------------*\
  =========                 |
  \\      /  F ield         | OpenFOAM: The Open Source CFD Toolbox
   \\    /   O peration     |
    \\  /    A nd           | www.openfoam.com
     \\/     M anipulation  |
-------------------------------------------------------------------------------
    Copyright (C) 2011-2016 OpenFOAM Foundation
    Copyright (C) 2017-2022 OpenCFD Ltd.
    Copyright (C) 2022-2023 Cineca
-------------------------------------------------------------------------------
License
    This file is part of OpenFOAM.

    OpenFOAM is free software: you can redistribute it and/or modify it
    under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    OpenFOAM is distributed in the hope that it will be useful, but WITHOUT
    ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
    FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License
    for more details.

    You should have received a copy of the GNU General Public License
    along with OpenFOAM.  If not, see <http://www.gnu.org/licenses/>.

\*---------------------------------------------------------------------------*/

// ************************************************************************* //

#include "cudaCoupledCsrMatrixExecutor.H"

#include "scalar.H"
#include "csrMatrix.H"
#include "global.cuh"
#include <hipcub/hipcub.hpp>

// * * * * * * * * * * * * * * * * CUDA Kernels  * * * * * * * * * * * * * * //

__global__
void cudaInitializeValueD
(
	const int   nBlocks,
    const int   nCells,
	const int   nOffsets,
	const int * const offsets,
	const int * const ldu2csr,
    const double * const diag,
          double * values
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i < nCells)
    {
    	for(int j=0; j<nOffsets;j++)
    	{
            values[ldu2csr[i]*nBlocks*nBlocks + offsets[j]] = diag[i*nOffsets + j];
    	}
    }
}

__global__
void cudaInitializeValueUL
(
	const int   nBlocks,
    const int   nCells,
    const int   nIntFaces,
	const int   nOffsets,
	const int * const offsets,
	const int * const ldu2csr,
    const double * const upper,
    const double * const lower,
          double * values
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i < nIntFaces)
    {
    	for(int j=0; j<nOffsets;j++)
    	{
            values[nBlocks*nBlocks*ldu2csr[nCells + i]+offsets[j]] =
                upper[i*nOffsets + j];
            values[nBlocks*nBlocks*ldu2csr[nCells + nIntFaces + i]+offsets[j]] =
                lower[i*nOffsets + j];
    	}
    }
}

__global__
void cudaInitializeValueExt
(
    const int   nCells,
    const int   nIntFaces,
    const int   nnzExt,
    const double * const extValues,
          double * valuesTmp
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i < nnzExt)
    {
        valuesTmp[nCells + 2*nIntFaces + i] = extValues[i];
    }
}

__global__
void cudaApplyValuePermutation 
(
    const int      length,
    const int      blockLen,
    const int    * const permArray,
    const double * const srcArray,
          double * dstArray
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i < length)
    {
        dstArray[permArray[i]] = srcArray[i];
    }
} 
//NOTA: this function (when csrAdressing will be joined back to csrMatrix) will 
//      become e template on the array type to be used both for adressing and 
//      values permutaiton

template <int nComps>
__global__
void cudaFillField
(
	const int nCells,
	const int position,
	const int nBlocks,
	const double * const input,
		  double * output
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i < nCells)
    {
    	for(int j=0; j<nComps; j++)
    	{
            output[i*nBlocks + position + j] = input[nComps*i + j];
    	}
    }
}

// * * * * * * * * * * * * * *  Wrapper functions * * * * * * * * * * * * * * //

template<class Type>
void Foam::cudaCoupledCsrMatrixExecutor::fillField
(
	const label nCells,
	const label position,
	const label nBlocks,
	const Field<Type>& input,
		  scalar * output
) const
{
    const label nComps = pTraits<Type>::nComponents;

	void* inPtr;
    label err = CHECK_CUDA_ERROR(hipMalloc((void**)&inPtr, (size_t) nCells*nBlocks*sizeof(scalar)));
    if (err != 0)
    {
        FatalErrorInFunction << "ERROR: hipMalloc returned " << err << abort(FatalError);
    }

    err = CHECK_CUDA_ERROR(hipMemcpy(inPtr, reinterpret_cast<const scalar*>(input.cdata()),
    		                             (size_t) nCells*nBlocks*sizeof(scalar), hipMemcpyHostToDevice));
    if (err != 0)
    {
        FatalErrorInFunction << "ERROR: hipMemcpy returned " << err << abort(FatalError);
    }

    int numBlocks = (nCells + NUM_THREADS_PER_BLOCK - 1) / NUM_THREADS_PER_BLOCK;
    cudaFillField<nComps><<<numBlocks, NUM_THREADS_PER_BLOCK>>>
    (
    	nCells,
		position,
		nBlocks,
		static_cast<const scalar*>(inPtr),
		output
    );

    hipDeviceSynchronize();
    CHECK_LAST_CUDA_ERROR();
    err = CHECK_CUDA_ERROR(hipFree(inPtr));
    CHECK_LAST_CUDA_ERROR();
}



void Foam::cudaCoupledCsrMatrixExecutor::initializeAndApplyValue
(
	const label nBlocks,
    const label nCells,
    const label nIntFaces,
    const label nOffsets,
    const label * const offsets,
    const label  * const ldu2csr,
    const scalar * const diag,
    const scalar * const upper,
    const scalar * const lower,
          scalar * values
) const
{
    label numBlocks;
    
    // Initialize valuesTmp = [(diag), (upper), (lower)]
    if (diag)
    {
        numBlocks = (nCells + NUM_THREADS_PER_BLOCK - 1) / NUM_THREADS_PER_BLOCK;
        cudaInitializeValueD<<<numBlocks, NUM_THREADS_PER_BLOCK>>>
        (
            nBlocks,
            nCells,
            nOffsets,
	    	offsets,
	    	ldu2csr,
            diag,
            values
        );
    }
    if (upper)
    {
        numBlocks = (nIntFaces + NUM_THREADS_PER_BLOCK - 1) / NUM_THREADS_PER_BLOCK;
        cudaInitializeValueUL<<<numBlocks, NUM_THREADS_PER_BLOCK>>>
        (
            nBlocks,
            nCells,
            nIntFaces,
	    	nOffsets,
	    	offsets,
	    	ldu2csr,
            upper,
            lower,
            values
        );
    }
    hipDeviceSynchronize();

    CHECK_LAST_CUDA_ERROR();
    return;
}


//void Foam::cudaCoupledCsrMatrixExecutor::initializeValueExt
//(
//    const label    nCells,
//    const label    nIntFaces,
//    const label    nnzExt,
//    const scalar * const diag,
//    const scalar * const upper,
//    const scalar * const lower,
//    const scalar * const extValue,
//          scalar * valuesTmp
//) const
//{
//    // Initialize valuesTmp = [(diag), (upper), (lower), (extValues)]
//    initializeValue
//    (
//        nCells,
//        nIntFaces,
//        diag,
//        upper,
//        lower,
//        valuesTmp
//    );
//
//    int numBlocks = (nnzExt + NUM_THREADS_PER_BLOCK - 1) / NUM_THREADS_PER_BLOCK;
//    cudaInitializeValueExt<<<numBlocks, NUM_THREADS_PER_BLOCK>>>
//    (
//        nCells,
//        nIntFaces,
//        nnzExt,
//        extValue,
//        valuesTmp
//    );
//
//    hipDeviceSynchronize();
//
//    CHECK_LAST_CUDA_ERROR();
//    return;
//}
//
//
//void Foam::cudaCoupledCsrMatrixExecutor::applyValuePermutation
//(
//    const label    totNnz,
//    const label  * const ldu2csr,
//    const scalar * const valuesTmp,
//          scalar * values,
//    const label    nBlocks
//) const
//{
//    int blockLen = nBlocks*nBlocks;
//    int numBlocks = (totNnz + NUM_THREADS_PER_BLOCK - 1) / NUM_THREADS_PER_BLOCK;
//    cudaApplyValuePermutation<<<numBlocks, NUM_THREADS_PER_BLOCK>>>
//    (
//        totNnz,
//        blockLen,
//        ldu2csr,
//        valuesTmp,
//        values
//    );
//
//    hipDeviceSynchronize();
//
//    CHECK_LAST_CUDA_ERROR();
//    return;
//}

// * * * * * * * * * * * * * Explicit instantiations  * * * * * * * * * * *  //

#define makecudaCoupledCsrMatrixExecutor(Type)                                 \
    template void Foam::cudaCoupledCsrMatrixExecutor::fillField<Type>          \
    (                                                                         \
	const Foam::label nCells,                                                       \
	const Foam::label position,                                                     \
	const Foam::label nBlocks,                                                      \
	const Foam::Field<Type>& input,                                               \
		  Foam::scalar * output                                                     \
    ) const;

makecudaCoupledCsrMatrixExecutor(Foam::scalar)
makecudaCoupledCsrMatrixExecutor(Foam::vector)
// ************************************************************************* //
