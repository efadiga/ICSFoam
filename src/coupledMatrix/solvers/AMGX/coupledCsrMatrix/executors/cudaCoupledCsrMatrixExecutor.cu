#include "hip/hip_runtime.h"
/*---------------------------------------------------------------------------*\
  =========                 |
  \\      /  F ield         | OpenFOAM: The Open Source CFD Toolbox
   \\    /   O peration     |
    \\  /    A nd           | www.openfoam.com
     \\/     M anipulation  |
-------------------------------------------------------------------------------
    Copyright (C) 2011-2016 OpenFOAM Foundation
    Copyright (C) 2017-2022 OpenCFD Ltd.
    Copyright (C) 2022-2023 Cineca
-------------------------------------------------------------------------------
License
    This file is part of OpenFOAM.

    OpenFOAM is free software: you can redistribute it and/or modify it
    under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    OpenFOAM is distributed in the hope that it will be useful, but WITHOUT
    ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
    FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License
    for more details.

    You should have received a copy of the GNU General Public License
    along with OpenFOAM.  If not, see <http://www.gnu.org/licenses/>.

\*---------------------------------------------------------------------------*/

// ************************************************************************* //

#include "cudaCoupledCsrMatrixExecutor.H"

#include "scalar.H"
#include "csrMatrix.H"
#include "global.cuh"
#include <hipcub/hipcub.hpp>

// * * * * * * * * * * * * * * * * CUDA Kernels  * * * * * * * * * * * * * * //

__global__
void cudaInitializeValueD
(
	const int   nBlocks,
    const int   nCells,
	const int   nOffsets,
	const int * const offsets,
	const int * const ldu2csr,
    const double * const diag,
          double * values
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i < nCells)
    {
    	for(int j=0; j<nOffsets;j++)
    	{
            values[ldu2csr[i]*nBlocks*nBlocks + offsets[j]] = diag[i*nOffsets + j];
    	}
    }
}

__global__
void cudaInitializeValueUL
(
	const int   nBlocks,
    const int   nCells,
    const int   nIntFaces,
	const int   nOffsets,
	const int * const offsets,
	const int * const ldu2csr,
    const double * const upper,
    const double * const lower,
          double * values
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i < nIntFaces)
    {
    	for(int j=0; j<nOffsets;j++)
    	{
            values[nBlocks*nBlocks*ldu2csr[nCells + i]+offsets[j]] =
                upper[i*nOffsets + j];
            values[nBlocks*nBlocks*ldu2csr[nCells + nIntFaces + i]+offsets[j]] =
                lower[i*nOffsets + j];
    	}
    }
}

__global__
void cudaInitializeValueExt
(
	const int   nBlocks,
    const int   nCells,
    const int   nIntFaces,
    const int   nnzExt,
	const int   nOffsets,
	const int * const offsets,
	const int * const ldu2csr,
    const double * const extValues,
          double * values
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i < nnzExt)
    {
    	for(int j=0; j<nOffsets;j++)
    	{
            values[nBlocks*nBlocks*ldu2csr[nCells + 2*nIntFaces + i]+offsets[j]] =
                extValues[i*nOffsets + j];
    	}
        //valuesTmp[nCells + 2*nIntFaces + i] = extValues[i];
    }
}

__global__
void cudaApplyValuePermutation 
(
    const int      length,
    const int      blockLen,
    const int    * const permArray,
    const double * const srcArray,
          double * dstArray
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i < length)
    {
        dstArray[permArray[i]] = srcArray[i];
    }
} 
//NOTA: this function (when csrAdressing will be joined back to csrMatrix) will 
//      become e template on the array type to be used both for adressing and 
//      values permutaiton

template <int nComps>
__global__
void cudaFillField
(
	const int nCells,
	const int position,
	const int nBlocks,
	const double * const input,
		  double * output
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i < nCells)
    {
    	for(int j=0; j<nComps; j++)
    	{
            output[i*nBlocks + position + j] = input[nComps*i + j];
    	}
    }
}

// * * * * * * * * * * * * * *  Wrapper functions * * * * * * * * * * * * * * //

template<class Type>
void Foam::cudaCoupledCsrMatrixExecutor::concatenate
(
    label size,
    const Field<Type>& lst,
    scalar * ptr,
	label consDispl
) const
{
       const label nC = pTraits<Type>::nComponents;
       label err = CHECK_CUDA_ERROR(
                   hipMemcpy(&ptr[consDispl*nC], lst.cdata(), (size_t) size*sizeof(Type), hipMemcpyHostToDevice)
               );
       if (err != 0)
       {
           FatalErrorInFunction << "ERROR: hipMemcpy returned " << err << abort(FatalError);
       }
}

template<class Type>
void Foam::cudaCoupledCsrMatrixExecutor::fillField
(
	const label nCells,
	const label position,
	const label nBlocks,
	const Field<Type>& input,
		  scalar * output
) const
{
    const label nComps = pTraits<Type>::nComponents;

	void* inPtr;
    label err = CHECK_CUDA_ERROR(hipMalloc((void**)&inPtr, (size_t) nCells*nComps*sizeof(scalar)));
    if (err != 0)
    {
        FatalErrorInFunction << "ERROR: hipMalloc returned " << err << abort(FatalError);
    }

    err = CHECK_CUDA_ERROR(hipMemcpy(inPtr, reinterpret_cast<const scalar*>(input.cdata()),
    		                             (size_t) nCells*nComps*sizeof(scalar), hipMemcpyHostToDevice));
    if (err != 0)
    {
        FatalErrorInFunction << "ERROR: hipMemcpy returned " << err << abort(FatalError);
    }

    int numBlocks = (nCells + NUM_THREADS_PER_BLOCK - 1) / NUM_THREADS_PER_BLOCK;
    cudaFillField<nComps><<<numBlocks, NUM_THREADS_PER_BLOCK>>>
    (
    	nCells,
		position,
		nBlocks,
		static_cast<const scalar*>(inPtr),
		output
    );

    hipDeviceSynchronize();
    CHECK_LAST_CUDA_ERROR();

    err = CHECK_CUDA_ERROR(hipFree(inPtr));
}

void Foam::cudaCoupledCsrMatrixExecutor::initializeAndApplyValue
(
	const label nBlocks,
    const label nCells,
    const label nIntFaces,
	const label nnzExt,
    const label nOffsets,
    const label * const offsets,
    const label  * const ldu2csr,
    const scalar * const diag,
    const scalar * const upper,
    const scalar * const lower,
    const scalar * const ext,
          scalar * values
) const
{
    label numBlocks;
    
    // Initialize valuesTmp = [(diag), (upper), (lower)]
    if (diag)
    {
        numBlocks = (nCells + NUM_THREADS_PER_BLOCK - 1) / NUM_THREADS_PER_BLOCK;
        cudaInitializeValueD<<<numBlocks, NUM_THREADS_PER_BLOCK>>>
        (
            nBlocks,
            nCells,
            nOffsets,
	    	offsets,
	    	ldu2csr,
            diag,
            values
        );
    CHECK_LAST_CUDA_ERROR();
    }
    if (upper)
    {
        numBlocks = (nIntFaces + NUM_THREADS_PER_BLOCK - 1) / NUM_THREADS_PER_BLOCK;
    	if (!lower)
    	{
            cudaInitializeValueUL<<<numBlocks, NUM_THREADS_PER_BLOCK>>>
            (
                nBlocks,
                nCells,
                nIntFaces,
                nOffsets,
                offsets,
                ldu2csr,
                upper,
                upper,
                values
            );
    CHECK_LAST_CUDA_ERROR();
    	}
    	else
    	{
            cudaInitializeValueUL<<<numBlocks, NUM_THREADS_PER_BLOCK>>>
            (
                nBlocks,
                nCells,
                nIntFaces,
                nOffsets,
                offsets,
                ldu2csr,
                upper,
                lower,
                values
            );
    CHECK_LAST_CUDA_ERROR();
    	}
    }
    if (ext)
    {
        numBlocks = (nnzExt + NUM_THREADS_PER_BLOCK - 1) / NUM_THREADS_PER_BLOCK;
        cudaInitializeValueExt<<<numBlocks, NUM_THREADS_PER_BLOCK>>>
        (
            nBlocks,
            nCells,
            nIntFaces,
			nnzExt,
	    	nOffsets,
	    	offsets,
	    	ldu2csr,
			ext,
            values
        );
    }
    hipDeviceSynchronize();

    CHECK_LAST_CUDA_ERROR();
    return;
}


//void Foam::cudaCoupledCsrMatrixExecutor::initializeValueExt
//(
//    const label    nCells,
//    const label    nIntFaces,
//    const label    nnzExt,
//    const scalar * const diag,
//    const scalar * const upper,
//    const scalar * const lower,
//    const scalar * const extValue,
//          scalar * valuesTmp
//) const
//{
//    // Initialize valuesTmp = [(diag), (upper), (lower), (extValues)]
//    initializeValue
//    (
//        nCells,
//        nIntFaces,
//        diag,
//        upper,
//        lower,
//        valuesTmp
//    );
//
//    int numBlocks = (nnzExt + NUM_THREADS_PER_BLOCK - 1) / NUM_THREADS_PER_BLOCK;
//    cudaInitializeValueExt<<<numBlocks, NUM_THREADS_PER_BLOCK>>>
//    (
//        nCells,
//        nIntFaces,
//        nnzExt,
//        extValue,
//        valuesTmp
//    );
//
//    hipDeviceSynchronize();
//
//    CHECK_LAST_CUDA_ERROR();
//    return;
//}
//
//
//void Foam::cudaCoupledCsrMatrixExecutor::applyValuePermutation
//(
//    const label    totNnz,
//    const label  * const ldu2csr,
//    const scalar * const valuesTmp,
//          scalar * values,
//    const label    nBlocks
//) const
//{
//    int blockLen = nBlocks*nBlocks;
//    int numBlocks = (totNnz + NUM_THREADS_PER_BLOCK - 1) / NUM_THREADS_PER_BLOCK;
//    cudaApplyValuePermutation<<<numBlocks, NUM_THREADS_PER_BLOCK>>>
//    (
//        totNnz,
//        blockLen,
//        ldu2csr,
//        valuesTmp,
//        values
//    );
//
//    hipDeviceSynchronize();
//
//    CHECK_LAST_CUDA_ERROR();
//    return;
//}

// * * * * * * * * * * * * * Explicit instantiations  * * * * * * * * * * *  //

#define makecudaCoupledCsrMatrixExecutor(Type)                                 \
    template void Foam::cudaCoupledCsrMatrixExecutor::fillField<Type>          \
    (                                                                         \
	const Foam::label nCells,                                                       \
	const Foam::label position,                                                     \
	const Foam::label nBlocks,                                                      \
	const Foam::Field<Type>& input,                                               \
		  Foam::scalar * output                                                     \
    ) const;

makecudaCoupledCsrMatrixExecutor(Foam::scalar)
makecudaCoupledCsrMatrixExecutor(Foam::vector)
template void Foam::cudaCoupledCsrMatrixExecutor::concatenate<Foam::scalar>
(
    Foam::label globSize,
    const Foam::Field<Foam::scalar> & lst,
    Foam::scalar * ptr,
	Foam::label consDispl
) const;

template void Foam::cudaCoupledCsrMatrixExecutor::concatenate<Foam::vector>
(
    Foam::label globSize,
    const Foam::Field<Foam::vector> & lst,
    Foam::scalar * ptr,
	Foam::label consDispl
) const;

template void Foam::cudaCoupledCsrMatrixExecutor::concatenate<Foam::tensor>
(
    Foam::label globSize,
    const Foam::Field<Foam::tensor> & lst,
    Foam::scalar * ptr,
	Foam::label consDispl
) const;

// ************************************************************************* //
